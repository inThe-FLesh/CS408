
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
__global__ void kernel() { printf("Hello from GPU!\n"); }

int main() {
  std::cout << "Hello, World!" << std::endl;
  kernel<<<1, 1>>>();
  hipDeviceSynchronize();
  return 0;
}
