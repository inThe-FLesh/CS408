#include "hip/hip_runtime.h"
#include "SHA256-Cuda.cuh"
#include <cassert>
#include <cstdio>
#include <exception>

/*I had to move all of the functions from the other files
  into this file, otherwise it wouldn't build correctly*/

__global__ void sha() {
  // Here we would use strSizes and strArr to set these values
  // I have had to set them manually as it was not working.
  int strSize = 12;
  const char *str = "RedBlockBlue";

  uint32_t hArr[] = {0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
                     0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19};

  uint8_t *bits = string_to_binary(str, strSize);

  uint32_t *paddedBits = pad_binary(bits, strSize);

  add_length_bits(paddedBits, (strSize * 8));

  uint32_t *schedule;

  schedule = prepare_message_schedule(paddedBits);

  compute_hash(schedule, hArr);

  /*printf("%x%x%x%x%x%x%x%x ", hArr[0], hArr[1], hArr[2], hArr[3], hArr[4],
         hArr[5], hArr[6], hArr[7]);*/

  free(paddedBits);
  free(bits);
  free(schedule);

  return;
}

int main() {

  int NUM_BLOCKS = 64;
  int NUM_THREADS = 64;

  struct timeval start, end;

  hipDeviceReset();

  // Timer code adapted from
  // https://www.geeksforgeeks.org/measure-execution-time-with-high-precision-in-c-c/
  // start timer.
  gettimeofday(&start, NULL);

  // unsync the I/O of C and C++.
  std::ios_base::sync_with_stdio(false);

  sha<<<NUM_BLOCKS, NUM_THREADS>>>();

  hipDeviceSynchronize();

  gettimeofday(&end, NULL);

  double time_taken;

  time_taken = (end.tv_sec - start.tv_sec) * 1e6;
  time_taken = (time_taken + (end.tv_usec - start.tv_usec)) * 1e-6;

  int hashesPerSecond = (int)60 * (NUM_BLOCKS * NUM_THREADS) / time_taken;

  cout << "Execution Time: " << time_taken << " seconds" << endl;
  cout << "hashes/s: " << hashesPerSecond << endl;
}

__host__ char *createCharArr(string *strArr, int strArrSize) {
  string output;

  for (int i = 0; i < strArrSize; i++) {
    output += strArr[i];
  }

  char *outputChar = (char *)malloc(sizeof(char) * (output.length() + 1));
  strcpy(outputChar, output.c_str());

  return outputChar;
}

__host__ int *getPositions(string *strArr, int strArrSize) {

  int *positions = (int *)malloc(sizeof(int) * (strArrSize + 1));

  positions[0] = 0;
  positions[1] = strArr[0].length();

  for (int i = 1; i < strArrSize; i++) {
    positions[i + 1] = strArr[i].length() + positions[i];
  }

  return positions;
}

__device__ char *getString(char *str, int *positions, int index) {
  int position = positions[index];
  int length = (positions[index + 1] - position);
  char *outputStr = (char *)malloc(sizeof(char) * length);

  for (int i = position, j = 0; i < length; i++, j++) {
    // using i and j here as the output string has to start at 0
    // and str has to start from position
    outputStr[j] = str[i];
  }
  return outputStr;
}

/*Preprocessing code*/

// Convert the string into binary representation. 8 bits per character.
__device__ uint8_t *string_to_binary(const char *str, const int strLen) {
  // breaks up each word in the string to an 8-bit binary number and adds them
  // to  the array.
  uint8_t *bits = (uint8_t *)malloc(sizeof(uint8_t) * strLen);

  for (int i = 0; i < strLen; i++) {
    bits[i] = (uint8_t)str[i];
  }

  return bits;
}

__device__ uint32_t *pad_binary(uint8_t *bits, int size) {

  uint32_t *paddedBits = (uint32_t *)malloc(sizeof(uint32_t) * 16);

  // dividing by 8 to get the number of 8 bit chunks
  int index = size;
  int remainder = index % 4;
  index = index / 4;

  for (int i = 0, j = 0; j < index; i += 4, j++) {
    // multiply each by 2^n to move the binary number n places to the left
    paddedBits[j] = (bits[i] << 24) + (bits[i + 1] << 16) + (bits[i + 2] << 8) +
                    bits[i + 3];
  }

  int i = index * 4;
  uint8_t append = 0x80;

  // Switch statement to determine what goes in the last 32 bit block. This
  // ensures that messages with characters not divisible by four are not
  // shortened.

  // Had to replace the powers with their values as it did not work on the GPU
  switch (remainder) {
  case 1:
    paddedBits[index] = (bits[i] << 24) + (append << 16);
    break;

  case 2:
    paddedBits[index] = (bits[i] << 24) + (bits[i + 1] << 16) + (append << 8);
    break;

  case 3:
    paddedBits[index] =
        (bits[i] << 24) + (bits[i + 1] << 16) + (bits[i + 2] << 8) + append;
    break;

  default:
    paddedBits[index] = append << 24;
    break;
  }

  return paddedBits;
}

// Uses the last 64 bits in order to record the length of the original
// message.
__device__ void add_length_bits(uint32_t *paddedBits, int sizeBits) {
  uint64_t lengthBits = sizeBits;
  uint64_t divider = 0xFFFFFFFF00000000;

  // Divider to get each half of the 64 bit length by anding them left and
  // right.
  uint32_t leftBits = lengthBits & divider;
  uint32_t rightBits = lengthBits & (divider >> 32);

  paddedBits[14] = leftBits;
  paddedBits[15] = rightBits;
}

/*Computation code*/

__device__ void compute_hash(uint32_t *W, uint32_t *hArr) {

  // declaring variables for computation

  uint32_t a = 0x6a09e667;
  uint32_t b = 0xbb67ae85;
  uint32_t c = 0x3c6ef372;
  uint32_t d = 0xa54ff53a;
  uint32_t e = 0x510e527f;
  uint32_t f = 0x9b05688c;
  uint32_t g = 0x1f83d9ab;
  uint32_t h = 0x5be0cd19;

  uint32_t k[64] = {
      0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1,
      0x923f82a4, 0xab1c5ed5, 0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
      0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174, 0xe49b69c1, 0xefbe4786,
      0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
      0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147,
      0x06ca6351, 0x14292967, 0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
      0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85, 0xa2bfe8a1, 0xa81a664b,
      0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
      0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a,
      0x5b9cca4f, 0x682e6ff3, 0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
      0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

  // computation beginning

  for (int t = 0; t < 64; t++) {
    uint32_t T1 =
        (h + big_sigma_one(e) + choose(e, f, g) + k[t] + W[t]) % 4294967296;
    uint32_t T2 = (big_sigma_zero(a) + majority(a, b, c)) % 4294967296;
    h = g;
    g = f;
    f = e;
    e = (d + T1) % 4294967296;
    d = c;
    c = b;
    b = a;
    a = (T1 + T2) % 4294967296;
  }

  hArr[0] = a + hArr[0];
  hArr[1] = b + hArr[1];
  hArr[2] = c + hArr[2];
  hArr[3] = d + hArr[3];
  hArr[4] = e + hArr[4];
  hArr[5] = f + hArr[5];
  hArr[6] = g + hArr[6];
  hArr[7] = h + hArr[7];
}

__device__ uint32_t right_rotation(uint32_t bits, int n) {
  // modulo 32 to ensure that ot can't ever shift more places than there are
  // bits
  n = n % 32;

  // optimised here to do the rotate right in one instruction. Improved
  // hashrate by roughly 100,000 hashes per second

  uint32_t shiftedBits = (bits >> n) ^ (bits << (32 - n));
  // uint32_t rotatedBits = bits << (32 - n);
  return shiftedBits;
}

__device__ uint32_t *prepare_message_schedule(uint32_t *paddedBits) {

  uint32_t *schedule = (uint32_t *)malloc(sizeof(uint32_t) * 64);

  for (int i = 0; i < 16; i++) {
    schedule[i] = paddedBits[i];
  }

  for (int t = 16; t <= 63; t++) {
    schedule[t] = sigma_one(schedule[t - 2]) + schedule[t - 7] +
                  sigma_zero(schedule[t - 15]) + (schedule[t - 16]);
  }

  return schedule;
}

// The formula to derive any value for W at position T
/*void build_message_schedule(uint32_t *W) {
  for (int t = 16; t <= 63; t++) {
    W[t] = sigma_one(W[t - 2]) + W[t - 7] + sigma_zero(W[t - 15]) + (W[t -
16]);
  }
}*/

__device__ uint32_t sigma_zero(uint32_t bits) {
  uint32_t sigmaReturn =
      (right_rotation(bits, 7)) ^ (right_rotation(bits, 18)) ^ (bits >> 3);
  return sigmaReturn;
}

__device__ uint32_t sigma_one(uint32_t bits) {
  uint32_t sigmaReturn =
      (right_rotation(bits, 17)) ^ (right_rotation(bits, 19)) ^ (bits >> 10);
  return sigmaReturn;
}

__device__ uint32_t big_sigma_zero(uint32_t bits) {
  uint32_t sigmaReturn = (right_rotation(bits, 2)) ^
                         (right_rotation(bits, 13)) ^
                         (right_rotation(bits, 22));
  return sigmaReturn;
}

__device__ uint32_t big_sigma_one(uint32_t bits) {
  uint32_t sigmaReturn = (right_rotation(bits, 6)) ^
                         (right_rotation(bits, 11)) ^
                         (right_rotation(bits, 25));
  return sigmaReturn;
}

__device__ uint32_t choose(uint32_t x, uint32_t y, uint32_t z) {
  uint32_t chooseReturn = (x & y) ^ (~x & z);
  return chooseReturn;
}

__device__ uint32_t majority(uint32_t x, uint32_t y, uint32_t z) {
  uint32_t majorityReturn = (x & y) ^ (x & z) ^ (y & z);
  return majorityReturn;
}
