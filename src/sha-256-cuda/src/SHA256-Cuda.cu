#include "hip/hip_runtime.h"
//
// Created by ross on 27/10/23.
//
//

// upgraded implementation to no longer use bitset and use malloc to declare
// arrays. All arrays are now passed as pointers for efficiency

#include "SHA256-Cuda.cuh"
#include "computation.cuh"
#include "preprocessing.cuh"
#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <iterator>

__global__ void sha(uint8_t **bitsArr, int *strSizes) {
  uint32_t *paddedBits;
  int threadID = threadIdx.x;
  uint8_t *bits = bitsArr[threadID];
  int strSize = strSizes[threadID];

  uint32_t hArr[] = {0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
                     0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19};

  paddedBits = pad_binary(bits, strSize);

  add_length_bits(paddedBits, (strSize * 8));

  uint32_t *schedule;

  schedule = prepare_message_schedule(paddedBits);

  compute_hash(schedule, hArr);

  printf("hash: ");

  for (uint32_t h : hArr) {
    printf("%#010x\n", h);
  }

  free(paddedBits);
  free(bits);
  free(schedule);
}

int main() {
  string strArr[] = {"RedBlockBlue", "12345", "zorgLover123", "openSesame"};
  uint8_t **bits = (uint8_t **)malloc(sizeof(uint8_t *) * size(strArr));
  int *strSizes = (int *)malloc(sizeof(int) * size(strArr));
  int count = 0;

  for (int i = 0; i < size(strArr); i++) {
    bits[i] = string_to_binary(strArr[i]);
    strSizes[i] = size(strArr);
  }

  sha<<<1, 4>>>(bits, strSizes);

  // solution for timer found on stack overflow
  /*auto now = std::chrono::steady_clock::now;
  duration<long> executeTime = 1s;
  auto start = now();

  while ((now() - start) < executeTime) {
    sha<<<1, 4>>>(bits, strSizes);
    count += 4;
  }*/

  free(bits);
  free(strSizes);

  cout << "Hashes per second: " << dec << count << endl;
  // sha(strArr[0]);
}
