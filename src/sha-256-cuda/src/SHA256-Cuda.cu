#include "hip/hip_runtime.h"

#include "SHA256-Cuda.cuh"
#include "computation.cuh"
#include "preprocessing.cuh"
#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <iterator>
#include <sys/types.h>

__global__ void sha(uint8_t **bitsArr, int *strSizes) {
  uint32_t *paddedBits;
  int threadID = threadIdx.x;
  uint8_t *bits = bitsArr[threadID];
  int strSize = strSizes[threadID];

  uint32_t hArr[] = {0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
                     0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19};

  paddedBits = pad_binary(bits, strSize);

  add_length_bits(paddedBits, (strSize * 8));

  uint32_t *schedule;

  schedule = prepare_message_schedule(paddedBits);

  compute_hash(schedule, hArr);

  for (unsigned int i : hArr) {
    printf("hello");
    printf("%#010x", i);
  }

  printf("\n");

  free(paddedBits);
  free(bits);
  free(schedule);
}

int main() {
  string strArr[] = {"RedBlockBlue", "12345", "zorgLover123", "openSesame"};
  size_t bitsBytes = sizeof(uint8_t *) * size(strArr);
  size_t strSizesBytes = sizeof(int) * size(strArr);
  uint8_t **h_bits = (uint8_t **)malloc(bitsBytes);
  int *h_strSizes = (int *)malloc(strSizesBytes);
  uint8_t **d_bits;
  int *d_strSizes;
  int count = 0;

  for (int i = 0; i < size(strArr); i++) {
    h_bits[i] = string_to_binary(strArr[i]);
    h_strSizes[i] = size(strArr);
  }

  hipMalloc(&d_bits, sizeof(uint8_t *) * size(strArr));
  hipMalloc(&d_strSizes, sizeof(int) * size(strArr));

  hipMemcpy(d_bits, h_bits, bitsBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_strSizes, h_strSizes, strSizesBytes, hipMemcpyHostToDevice);

  free(h_bits);
  free(h_strSizes);

  sha<<<1, 4>>>(d_bits, d_strSizes);
  hipDeviceSynchronize();

  // solution for timer found on stack overflow
  /*auto now = std::chrono::steady_clock::now;
  duration<long> executeTime = 1s;
  auto start = now();

  while ((now() - start) < executeTime) {
    sha<<<1, 4>>>(bits, strSizes);
    count += 4;
  }*/

  cout << "Hashes per second: " << dec << count << endl;
  // sha(strArr[0]);
}
