#include "hip/hip_runtime.h"

#include "SHA256-Cuda.cuh"
#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <iterator>
#include <sys/types.h>

/*I had to move all of the functions from the other files
  into this file, otherwise it wouldn't build correctly*/

__global__ void sha(uint8_t **bitsArr, int *strSizes, uint32_t *hArr) {
    uint32_t  *paddedBits;
    int threadID = threadIdx.x;
    uint8_t *bits = bitsArr[threadID];
    int strSize = strSizes[threadID];
    uint32_t hArrOut[8];

    paddedBits = pad_binary(bits, strSize);

    add_length_bits(paddedBits, (strSize * 8));

    uint32_t* schedule;

    schedule = prepare_message_schedule(paddedBits);

    compute_hash(schedule, hArr);

    for (int i = 0; i < 8; i++) {
        hArrOut[i] = hArr[i];
    }

    printf("jello");

    free(paddedBits);
    free(bits);
    free(schedule);
}


int main() {
    string strArr[] = { "RedBlockBlue", "12345", "zorgLover123", "openSesame" };

    size_t bitsBytes = sizeof(uint8_t*) * size(strArr);
    size_t strSizesBytes = sizeof(int) * size(strArr);
    size_t hArrSizeBytes = sizeof(uint32_t) * 8;

    uint8_t** h_bits = (uint8_t**)malloc(bitsBytes);
    int* h_strSizes = (int*)malloc(strSizesBytes);

    uint8_t** d_bits;
    int* d_strSizes;
    uint32_t* d_hArr;
    
    int count = 0;


    uint32_t hArr[] = { 0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
                        0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19 };

    for (int i = 0; i < size(strArr); i++) {
        h_bits[i] = string_to_binary(strArr[i]);
        h_strSizes[i] = size(strArr[i]);
    }

    hipMalloc(&d_bits, bitsBytes);
    hipMalloc(&d_strSizes, strSizesBytes);
    hipMalloc(&d_hArr, hArrSizeBytes);

    hipMemcpy(d_bits, h_bits, bitsBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_strSizes, h_strSizes, strSizesBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_hArr, hArr, hArrSizeBytes, hipMemcpyHostToDevice);

    free(h_bits);
    free(h_strSizes);

    sha <<<1, 4 >>> (d_bits, d_strSizes, d_hArr);
    hipDeviceSynchronize();

    // solution for timer found on stack overflow
    /*auto now = std::chrono::steady_clock::now;
    duration<long> executeTime = 1s;
    auto start = now();

    while ((now() - start) < executeTime) {
      sha<<<1, 4>>>(bits, strSizes);
      count += 4;
    }*/

    cout << "Hashes per second: " << dec << count << endl;
    // sha(strArr[0]);
}


/*Preprocessing code*/

// Convert the string into binary representation. 8 bits per character.
__host__ uint8_t* string_to_binary(const string& str) {
    // breaks up each word in the string to an 8-bit binary number and adds them
    // to  the array.
    uint8_t* bits = (uint8_t*)malloc(sizeof(uint8_t) * str.size());

    for (int i = 0; i < str.length(); i++) {
        bits[i] = uint8_t(str[i]);
    }

    return bits;
}

__device__ uint32_t* pad_binary(uint8_t* bits, int size) {

    uint32_t* paddedBits = (uint32_t*)malloc(sizeof(uint32_t) * 16);

    // dividing by 8 to get the number of 8 bit chunks
    int index = size;
    int remainder = index % 4;
    index = index / 4;

    for (int i = 0, j = 0; j < index; i += 4, j++) {
        // multiply each by 2^n to move the binary number n places to the left
        paddedBits[j] = bits[i] * 16777216 + bits[i + 1] * 65536 +
            bits[i + 2] * 256 + bits[i + 3];
    }

    int i = index * 4;
    uint8_t append = 0x80;

    // Switch statement to determine what goes in the last 32 bit block. This
    // ensures that messages with characters not divisible by four are not
    // shortened.

    // Had to replace the powers with their values as it did not work on the GPU
    switch (remainder) {
    case 1:
        paddedBits[index] = bits[i] * 16777216 + append * 65536;
        break;

    case 2:
        paddedBits[index] =
            bits[i] * 1677721 + bits[i + 1] * 65536 + append * 256;
        break;

    case 3:
        paddedBits[index] = bits[i] * 1677721 + bits[i + 1] * 65536 +
            bits[i + 2] * 256 + append;
        break;

    default:
        paddedBits[index] = append * 1677721;
        break;
    }

    return paddedBits;
}

// Uses the last 64 bits in order to record the length of the original message.
__device__ void add_length_bits(uint32_t* paddedBits, int sizeBits) {
    uint64_t lengthBits = sizeBits;
    uint64_t divider = 0xFFFFFFFF00000000;

    // Divider to get each half of the 64 bit length by anding them left and
    // right.
    uint32_t leftBits = lengthBits & divider;
    uint32_t rightBits = lengthBits & (divider >> 32);

    paddedBits[14] = leftBits;
    paddedBits[15] = rightBits;
}


/*Computation code*/

__device__ void compute_hash(uint32_t* W, uint32_t* hArr) {

    // declaring variables for computation

    uint32_t a = 0x6a09e667;
    uint32_t b = 0xbb67ae85;
    uint32_t c = 0x3c6ef372;
    uint32_t d = 0xa54ff53a;
    uint32_t e = 0x510e527f;
    uint32_t f = 0x9b05688c;
    uint32_t g = 0x1f83d9ab;
    uint32_t h = 0x5be0cd19;

    uint32_t k[64] = {
        0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1,
        0x923f82a4, 0xab1c5ed5, 0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
        0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174, 0xe49b69c1, 0xefbe4786,
        0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
        0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147,
        0x06ca6351, 0x14292967, 0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
        0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85, 0xa2bfe8a1, 0xa81a664b,
        0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
        0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a,
        0x5b9cca4f, 0x682e6ff3, 0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
        0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2 };

    // computation beginning

    for (int t = 0; t < 64; t++) {
        uint32_t T1 =
            (h + big_sigma_one(e) + choose(e, f, g) + k[t] + W[t]) % 4294967296;
        uint32_t T2 = (big_sigma_zero(a) + majority(a, b, c)) % 4294967296;
        h = g;
        g = f;
        f = e;
        e = (d + T1) % 4294967296;
        d = c;
        c = b;
        b = a;
        a = (T1 + T2) % 4294967296;
    }

    hArr[0] = a + hArr[0];
    hArr[1] = b + hArr[1];
    hArr[2] = c + hArr[2];
    hArr[3] = d + hArr[3];
    hArr[4] = e + hArr[4];
    hArr[5] = f + hArr[5];
    hArr[6] = g + hArr[6];
    hArr[7] = h + hArr[7];
}

__device__ uint32_t right_rotation(uint32_t bits, int n) {
    // modulo 32 to ensure that ot can't ever shift more places than there are
    // bits
    n = n % 32;

    // optimised here to do the rotate right in one instruction. Improved hashrate
    // by roughly 100,000 hashes per second

    uint32_t shiftedBits = (bits >> n) ^ (bits << (32 - n));
    // uint32_t rotatedBits = bits << (32 - n);
    return shiftedBits;
}

__device__ uint32_t* prepare_message_schedule(uint32_t* paddedBits) {

    uint32_t* schedule = (uint32_t*)malloc(sizeof(uint32_t) * 64);

    for (int i = 0; i < 16; i++) {
        schedule[i] = paddedBits[i];
    }

    for (int t = 16; t <= 63; t++) {
        schedule[t] = sigma_one(schedule[t - 2]) + schedule[t - 7] +
            sigma_zero(schedule[t - 15]) + (schedule[t - 16]);
    }

    return schedule;
}

// The formula to derive any value for W at position T
/*void build_message_schedule(uint32_t *W) {
  for (int t = 16; t <= 63; t++) {
    W[t] = sigma_one(W[t - 2]) + W[t - 7] + sigma_zero(W[t - 15]) + (W[t - 16]);
  }
}*/

__device__ uint32_t sigma_zero(uint32_t bits) {
    uint32_t sigmaReturn =
        (right_rotation(bits, 7)) ^ (right_rotation(bits, 18)) ^ (bits >> 3);
    return sigmaReturn;
}

__device__ uint32_t sigma_one(uint32_t bits) {
    uint32_t sigmaReturn =
        (right_rotation(bits, 17)) ^ (right_rotation(bits, 19)) ^ (bits >> 10);
    return sigmaReturn;
}

__device__ uint32_t big_sigma_zero(uint32_t bits) {
    uint32_t sigmaReturn = (right_rotation(bits, 2)) ^
        (right_rotation(bits, 13)) ^
        (right_rotation(bits, 22));
    return sigmaReturn;
}

__device__ uint32_t big_sigma_one(uint32_t bits) {
    uint32_t sigmaReturn = (right_rotation(bits, 6)) ^
        (right_rotation(bits, 11)) ^
        (right_rotation(bits, 25));
    return sigmaReturn;
}

__device__ uint32_t choose(uint32_t x, uint32_t y, uint32_t z) {
    uint32_t chooseReturn = (x & y) ^ (~x & z);
    return chooseReturn;
}

__device__ uint32_t majority(uint32_t x, uint32_t y, uint32_t z) {
    uint32_t majorityReturn = (x & y) ^ (x & z) ^ (y & z);
    return majorityReturn;
}
