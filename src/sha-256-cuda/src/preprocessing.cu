#include "hip/hip_runtime.h"
//
// Created by ross on 28/10/23.
//

#include "preprocessing.cuh"
#include <cstdint>

// Convert the string into binary representation. 8 bits per character.
uint8_t *string_to_binary(const string &str) {
  // breaks up each word in the string to an 8-bit binary number and adds them
  // to  the array.
  uint8_t *bits = (uint8_t *)malloc(sizeof(uint8_t) * str.size());

  for (int i = 0; i < str.length(); i++) {
    bits[i] = uint8_t(str[i]);
  }

  return bits;
}

// divides the characters into 32 bit blocks then appends a single one and pads
// with zeros.
uint32_t *pad_binary(uint8_t *bits, int size) {

  uint32_t *paddedBits = (uint32_t *)malloc(sizeof(uint32_t) * 16);

  // dividing by 8 to get the number of 8 bit chunks
  int index = size;
  int remainder = index % 4;
  index = index / 4;

  for (int i = 0, j = 0; j < index; i += 4, j++) {
    // multiply each by 2^n to move the binary number n places to the left
    paddedBits[j] = bits[i] * pow(2, 24) + bits[i + 1] * pow(2, 16) +
                    bits[i + 2] * pow(2, 8) + bits[i + 3];
  }

  int i = index * 4;
  uint8_t append = 0x80;

  // Switch statement to determine what goes in the last 32 bit block. This
  // ensures that messages with characters not divisible by four are not
  // shortened.
  switch (remainder) {
  case 1:
    paddedBits[index] = bits[i] * pow(2, 24) + append * pow(2, 16);
    break;

  case 2:
    paddedBits[index] =
        bits[i] * pow(2, 24) + bits[i + 1] * pow(2, 16) + append * pow(2, 8);
    break;

  case 3:
    paddedBits[index] = bits[i] * pow(2, 24) + bits[i + 1] * pow(2, 16) +
                        bits[i + 2] * pow(2, 8) + append;
    break;

  default:
    paddedBits[index] = append * pow(2, 24);
    break;
  }

  return paddedBits;
}

// Uses the last 64 bits in order to record the length of the original message.
void add_length_bits(uint32_t *paddedBits, int sizeBits) {
  uint64_t lengthBits = sizeBits;
  uint64_t divider = 0xFFFFFFFF00000000;

  // Divider to get each half of the 64 bit length by anding them left and
  // right.
  uint32_t leftBits = lengthBits & divider;
  uint32_t rightBits = lengthBits & (divider >> 32);

  paddedBits[14] = leftBits;
  paddedBits[15] = rightBits;
}
