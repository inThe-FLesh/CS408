//
// Created by ross on 27/10/23.
//

#include "headers/sha256Cuda.h"

void sha(const string& str) {

    // variable declarations
    bitset<8> bits[str.size()];

    u_int32_t hArr[] = {0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
                    0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19};

    // Malloc variables for kernel calls
    int scheduleSize = sizeof(u_int32_t[64]);
    int paddedBitsSize = sizeof(u_int32_t[16]);

    auto* h_paddedBits = (u_int32_t*)malloc(paddedBitsSize);
    auto* h_schedule = (u_int32_t*)malloc(scheduleSize);
    auto* h_count = (int*)malloc(sizeof(int));

    *h_count = 1;

    u_int32_t *d_paddedBits;
    hipMalloc(&d_paddedBits, paddedBitsSize);

    u_int32_t *d_schedule;
    hipMalloc(&d_schedule, scheduleSize);

    int *d_count;
    hipMalloc(&d_count, sizeof(int));

    string_to_binary(str, bits);

    pad_binary(h_paddedBits, bits, sizeof(bits));

    add_length_bits(h_paddedBits, sizeof(bits));

    // Calls to kernel functions

    hipMemcpy(d_paddedBits, h_paddedBits, paddedBitsSize, hipMemcpyHostToDevice);
    prepare_message_schedule<<<1, 16>>>(d_schedule, d_paddedBits);
    hipMemcpy(h_schedule, d_schedule, scheduleSize, hipMemcpyDeviceToHost);

    build_message_schedule(h_schedule);

    compute_hash(h_schedule, hArr);

    free(h_schedule);
    free(h_paddedBits);
    free(h_count);

    hipFree(d_schedule);
    hipFree(d_paddedBits);

    for (u_int32_t h : hArr) {
        cout << setfill('0') << hex << setw(8) << h;
    }

    cout << endl;
}

int main() {
    string strArr[] = {"RedBlockBlue", "12345", "zorgLover123"};
    //int count = 0;

    // solution for timer found on stack overflow
    /*auto now = std::chrono::steady_clock::now;
    duration<long> executeTime = 1s;
    auto start = now();

    while ((now() - start) < executeTime) {
      for (const string& str : strArr) {
        sha(str);
        count += 1;
      }
    }

    cout << "Hashes per second: " << dec << count << endl;*/

    sha(strArr[0]);
}
