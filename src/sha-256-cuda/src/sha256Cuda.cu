//
// Created by ross on 27/10/23.
//

<<<<<<< HEAD
#include "computation.cuh"
#include "sha256Cuda.cuh"

void sha(string *strArr) {

  // defining 2D arrays using malloc
  bitset<32>(*schedules)[64] =
      (bitset<32>(*)[64])malloc(sizeof(bitset<32>[64][64]));

  uint32_t(*hArrs)[64] = (uint32_t(*)[64])malloc(sizeof(uint32_t[8][64]));

  for (int i = 0; i < 64; i++) {
    string str = strArr[i];

    bitset<8> bits[str.size()];
    bitset<32> paddedBits[16];

    uint32_t hArr[] = {0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
                       0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19};

    for (int j = 0; j < 8; j++) {
      hArrs[j][i] = hArr[j];
    }

    string_to_binary(str, bits);

    pad_binary(paddedBits, bits, sizeof(bits));

    /*for (bitset<32> pad : paddedBits){
        cout << pad << endl;
    }*/

    add_length_bits(paddedBits, sizeof(bits));

    bitset<32> schedule[64];

    prepare_message_schedule(schedule, paddedBits);

    for (int j = 0; j < 64; j++) {
      schedules[j][i] = hArr[j];
    }
  }

  /*for (ulong h : hArr) {
    cout << setfill('0') << hex << setw(8) << h;
  }

  cout << endl;*/
}

int main() {
  string *strArr = (string *)malloc(sizeof(string) * 64);
  int count = 0;

  // solution for timer found on stack overflow
  auto now = std::chrono::steady_clock::now;
  duration<long> executeTime = 1s;
  auto start = now();

  while ((now() - start) < executeTime) {
  }

  cout << "Hashes per second: " << dec << count << endl;
=======
#include "headers/sha256Cuda.h"

void sha(const string& str) {

    // variable declarations
    bitset<8> bits[str.size()];

    u_int32_t hArr[] = {0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
                    0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19};

    // Malloc variables for kernel calls
    int scheduleSize = sizeof(u_int32_t[64]);
    int paddedBitsSize = sizeof(u_int32_t[16]);

    auto* h_paddedBits = (u_int32_t*)malloc(paddedBitsSize);
    auto* h_schedule = (u_int32_t*)malloc(scheduleSize);
    auto* h_count = (int*)malloc(sizeof(int));

    *h_count = 1;

    u_int32_t *d_paddedBits;
    hipMalloc(&d_paddedBits, paddedBitsSize);

    u_int32_t *d_schedule;
    hipMalloc(&d_schedule, scheduleSize);

    int *d_count;
    hipMalloc(&d_count, sizeof(int));

    string_to_binary(str, bits);

    pad_binary(h_paddedBits, bits, sizeof(bits));

    add_length_bits(h_paddedBits, sizeof(bits));

    // Calls to kernel functions

    hipMemcpy(d_paddedBits, h_paddedBits, paddedBitsSize, hipMemcpyHostToDevice);
    prepare_message_schedule<<<1, 16>>>(d_schedule, d_paddedBits);
    hipMemcpy(h_schedule, d_schedule, scheduleSize, hipMemcpyDeviceToHost);

    build_message_schedule(h_schedule);

    compute_hash(h_schedule, hArr);

    free(h_schedule);
    free(h_paddedBits);
    free(h_count);

    hipFree(d_schedule);
    hipFree(d_paddedBits);

    for (u_int32_t h : hArr) {
        cout << setfill('0') << hex << setw(8) << h;
    }

    cout << endl;
}

int main() {
    string strArr[] = {"RedBlockBlue", "12345", "zorgLover123"};
    //int count = 0;

    // solution for timer found on stack overflow
    /*auto now = std::chrono::steady_clock::now;
    duration<long> executeTime = 1s;
    auto start = now();

    while ((now() - start) < executeTime) {
      for (const string& str : strArr) {
        sha(str);
        count += 1;
      }
    }

    cout << "Hashes per second: " << dec << count << endl;*/

    sha(strArr[0]);
>>>>>>> refs/remotes/origin/dev
}
